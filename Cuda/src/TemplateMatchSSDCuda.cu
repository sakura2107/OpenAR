#include "hip/hip_runtime.h"
#include <TemplateMatchSSDCuda.cuh>
#include <hip/hip_runtime.h>
#include <>
#include <random>
#include <stdio.h>

__global__ void templateMatchSSDKernel
(const unsigned char* image, 
 const unsigned char* temp, 
 size_t* result, 
 int* compare_points, 
 int image_width, 
 int image_height, 
 int temp_width, 
 int temp_height,
 int num_points)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x + temp_width <= image_width && y + temp_height <= image_height) {
        size_t sum = 0;
        for (int i = 0; i < num_points; i++) {
            int temp_x = compare_points[i * 2];
            int temp_y = compare_points[i * 2 + 1];
            int image_x = x + temp_x;
            int image_y = y + temp_y;
            if (image_x < image_width && image_y < image_height) { 
                size_t diff = (image[image_y * image_width + image_x] - temp[temp_y * temp_width + temp_x]);
                sum += diff * diff;
            }
        }
        result[y * (image_width - temp_width + 1) + x] = sum;
    }
}

bool CUDAMATCH_API ar::templateMatchSSDCuda(int* res,
    std::string& msg_res,
    const unsigned char* image,
    const unsigned char* temp,
    const int& image_width,
    const int& image_height,
    const int& temp_width,
    const int& temp_height,
    const float& threshold,
    const int& num_points) 
{
    hipError_t err = hipSuccess;
    msg_res = "Success";

    const int image_size = image_width * image_height;
    const int temp_size = temp_width * temp_height;
    const int result_size = (image_width - temp_width + 1) * (image_height - temp_height + 1);

    size_t* result_array = new size_t[result_size];
    //x_0, y_0, x_1, y_1, x_2, y_2, ...
    int* compare_points_array = new int[num_points * 2]; 

    unsigned char* device_image_array = nullptr;
    unsigned char* device_temp_array = nullptr;
    size_t* device_result_array = nullptr;
    int* device_compare_points_array = nullptr;

    makePointsRandom(compare_points_array, num_points, temp_width, temp_height);

    err = hipMalloc(&device_image_array, image_size * sizeof(unsigned char));
    if (err != hipSuccess) {
        delete[] result_array;
        delete[] compare_points_array;
        msg_res = hipGetErrorString(err);
        return false;
    }
    err = hipMalloc(&device_temp_array, temp_size * sizeof(unsigned char));
    if (err != hipSuccess) {
        delete[] result_array;
        delete[] compare_points_array;
        hipFree(device_image_array);
        msg_res = hipGetErrorString(err);
        return false;
    }
    err = hipMalloc(&device_result_array, result_size * sizeof(size_t));
    if (err != hipSuccess) {
        delete[] result_array;
        delete[] compare_points_array;
        hipFree(device_image_array);
        hipFree(device_temp_array);
        msg_res = hipGetErrorString(err);
        return false;
    }
    err = hipMalloc(&device_compare_points_array, num_points * 2 * sizeof(int));
    if (err != hipSuccess) {
        delete[] result_array;
        delete[] compare_points_array;
        hipFree(device_image_array);
        hipFree(device_temp_array);
        hipFree(device_result_array);
        msg_res = hipGetErrorString(err);
        return false;
    }

    err = hipMemcpy(device_image_array, image, image_size * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        delete[] result_array;
        delete[] compare_points_array;
        hipFree(device_image_array);
        hipFree(device_temp_array);
        hipFree(device_result_array);
        hipFree(device_compare_points_array);
        msg_res = hipGetErrorString(err);
        return false;
    }
    err = hipMemcpy(device_temp_array, temp, temp_size * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        delete[] result_array;
        delete[] compare_points_array;
        hipFree(device_image_array);
        hipFree(device_temp_array);
        hipFree(device_result_array);
        hipFree(device_compare_points_array);
        msg_res = hipGetErrorString(err);
        return false;
    }
    err = hipMemcpy(device_compare_points_array, compare_points_array, num_points * 2 * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        delete[] result_array;
        delete[] compare_points_array;
        hipFree(device_image_array);
        hipFree(device_temp_array);
        hipFree(device_result_array);
        hipFree(device_compare_points_array);
        msg_res = hipGetErrorString(err);
        return false;
    }


    dim3 threadsPerBlock(64, 64);
    dim3 blocksPerGrid((image_width + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (image_height + threadsPerBlock.y - 1) / threadsPerBlock.y);
    templateMatchSSDKernel << <blocksPerGrid, threadsPerBlock >> > (device_image_array,
        device_temp_array,
        device_result_array,
        device_compare_points_array,
        image_width,
        image_height,
        temp_width,
        temp_height,
        num_points);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        delete[] result_array;
        delete[] compare_points_array;
        hipFree(device_image_array);
        hipFree(device_temp_array);
        hipFree(device_result_array);
        hipFree(device_compare_points_array);
        msg_res = hipGetErrorString(err);
        return false;
    }

    err = hipMemcpy(result_array, device_result_array, result_size * sizeof(size_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        delete[] result_array;
        delete[] compare_points_array;
        hipFree(device_image_array);
        hipFree(device_temp_array);
        hipFree(device_result_array);
        hipFree(device_compare_points_array);
        msg_res = hipGetErrorString(err);
        return false;
    }

    size_t cur_min_ssd = (size_t)255 * 255 * num_points;
    for (int i = 0; i < result_size; i++) {
        if (result_array[i] < (size_t)2000 * (1 - threshold) * num_points && result_array[i] < cur_min_ssd)
        {
            printf("%d", result_array[i]);
            int x = i % (image_width - temp_width + 1);
            int y = i / (image_width - temp_width + 1);
            res[0] = x;
            res[1] = y;
            cur_min_ssd = result_array[i];
        }
    }

    err = hipFree(device_image_array);
    if (err != hipSuccess) {
        delete[] result_array;
        delete[] compare_points_array;
        hipFree(device_temp_array);
        hipFree(device_result_array);
        hipFree(device_compare_points_array);
        msg_res = hipGetErrorString(err);
        return false;
    }
    err = hipFree(device_temp_array);
    if (err != hipSuccess) {
        delete[] result_array;
        delete[] compare_points_array;
        hipFree(device_result_array);
        hipFree(device_compare_points_array);
        msg_res = hipGetErrorString(err);
        return false;
    }
    err = hipFree(device_result_array);
    if (err != hipSuccess) {
        delete[] result_array;
        delete[] compare_points_array;
        hipFree(device_compare_points_array);
        msg_res = hipGetErrorString(err);
        return false;
    }
    err = hipFree(device_compare_points_array);
    if (err != hipSuccess) {
        delete[] result_array;
        delete[] compare_points_array;
        msg_res = hipGetErrorString(err);
        return false;
    }
    delete[] result_array;
    delete[] compare_points_array;

    return true;
}

void ar::makePointsRandom(int* division_random_point, const int& num_points, const int& temp_width, const int& temp_height) {
    int* best_division = new int[2];
    ar::findBestDivision(best_division, num_points);

    int region_height = best_division[0];
    int region_width = best_division[1];

    int height_per_region = temp_height / region_height;
    int width_per_region = temp_width / region_width;

    std::random_device rd;
    std::mt19937 generator(rd());

    for (int i = 0; i < region_height; i++) {
        for (int j = 0; j < region_width; j++) {
            std::uniform_int_distribution<int> distribution_x(j * width_per_region, (j + 1) * width_per_region - 1);
            int x_ = distribution_x(generator);
            std::uniform_int_distribution<int> distribution_y(i * height_per_region, (i + 1) * height_per_region - 1);
            int y_ = distribution_y(generator);
            division_random_point[(i * region_width + j) * 2] = x_;
            division_random_point[(i * region_width + j) * 2 + 1] = y_;
        }
    }
    delete[] best_division;
}

void ar::findBestDivision(int* best_division, const int& num_points) {
    for (int i = std::sqrt(num_points); i >= 1; i--) {
        if (num_points % i == 0) {
            best_division[0] = i;
            best_division[1] = num_points / i;
        }
    }
    best_division[0] = 1;
    best_division[1] = num_points;
}